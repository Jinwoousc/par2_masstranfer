#include "hip/hip_runtime.h"
/**
* @file PParticles.cu
* @brief Implementation file for PParticles class.
*
* @author Calogero B. Rizzo
*
* @copyright This file is part of the PAR2 software.
*            Copyright (C) 2018 Calogero B. Rizzo
*
* @license This program is free software: you can redistribute it and/or modify
*          it under the terms of the GNU General Public License as published by
*          the Free Software Foundation, either version 3 of the License, or
*          (at your option) any later version.
*
*          This program is distributed in the hope that it will be useful,
*          but WITHOUT ANY WARRANTY; without even the implied warranty of
*          MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*          GNU General Public License for more details.
*
*          You should have received a copy of the GNU General Public License
*          along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "../Geometry/CartesianGrid.cuh"
#include "../Geometry/FaceField.cuh"
#include "../Geometry/CornerField.cuh"
#include "../Geometry/MassField.cuh"

#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <fstream>
#include <algorithm>

namespace par2
{
    template<typename T>
    struct InitCURAND
    {
        unsigned long long seed;
        hiprandState_t *states;
        InitCURAND(unsigned long long _seed, thrust::device_vector<hiprandState_t> &_states)
        {
            seed = _seed;
            states = thrust::raw_pointer_cast(_states.data());
        }

        __device__
        void operator()(unsigned int i)
        {
            hiprand_init(seed, i, 0, &states[i]);
        }
    };

    template<typename T>
    struct InitVolume
    {
        hiprandState_t* states;
        grid::Grid<T> grid;
        int* mmb1;
        unsigned int nParticles;
        T p1x, p1y, p1z;
        T p2x, p2y, p2z;
        InitVolume(thrust::device_vector<hiprandState_t> &_states,
                    const grid::Grid<T> &_grid,
                    thrust::device_vector<int> &_mmb1,
                    unsigned int _nParticles,
                    T _p1x, T _p1y, T _p1z,
                    T _p2x, T _p2y, T _p2z)
        {
            states = thrust::raw_pointer_cast(_states.data());
            grid = _grid;
            mmb1 = thrust::raw_pointer_cast(_mmb1.data());
            nParticles = _nParticles;
            p1x = _p1x;
            p1y = _p1y;
            p1z = _p1z;
            p2x = _p2x;
            p2y = _p2y;
            p2z = _p2z;
        }

        using Position = thrust::tuple<T, T, T, int>;

        __device__
        Position operator()(unsigned int i) const
        {
            Position p;

            thrust::get<0>(p) = p1x + (p2x-p1x)*hiprand_uniform(&states[i]);
            thrust::get<1>(p) = p1y + (p2y-p1y)*hiprand_uniform(&states[i]);
            thrust::get<2>(p) = p1z + (p2z-p1z)*hiprand_uniform(&states[i]);
            thrust::get<3>(p) = 0;

            // heterogeneous masstransfer
            int idx, idy, idz;
            grid::idPoint(grid,
                          thrust::get<0>(p),
                          thrust::get<1>(p),
                          thrust::get<2>(p),
                          &idx, &idy, &idz);
            int id = grid::mergeId(grid, idx, idy, idz);
            atomicAdd( &mmb1[id], 1 );

            // homogeneous masstransfer
            // atomicAdd( &mmb1[0], 1 );

            return p;
        }
    };

    template<typename T>
    PParticles<T>::PParticles(const grid::Grid<T> &_grid,
                           thrust::host_vector<int> &_mmb1,
                           thrust::host_vector<int> &_mmb2,
                           thrust::host_vector<int> &_mimb1,
                           thrust::host_vector<int> &_mimb2,
                           const thrust::host_vector<T> &_datax,
                           const thrust::host_vector<T> &_datay,
                           const thrust::host_vector<T> &_dataz,
                           T _molecularDiffusion,
                           T _alphaL,
                           T _alphaT,
                           int _massType,
                           T _mass1alpha,
                           T _mass1beta,
                           T _mass2alpha,
                           T _mass2beta,
                           unsigned int _nParticles,
                           long int _seed,
                           bool _useTrilinearCorrection)
            : nParticles(_nParticles), molecularDiffusion(_molecularDiffusion),
              alphaL(_alphaL), alphaT(_alphaT), grid(_grid), moveParticle(_grid),
              useTrilinearCorrection(_useTrilinearCorrection), massType(_massType),
              mass1alpha(_mass1alpha), mass1beta(_mass1beta), mass2alpha(_mass2alpha), mass2beta(_mass2beta)
    {
        cx.resize(nParticles);
        cy.resize(nParticles);
        cz.resize(nParticles);
        cp.resize(nParticles);
        cm.resize(nParticles);

        mmb1 = _mmb1;
        mmb2 = _mmb2;
        mimb1 = _mimb1;
        mimb2 = _mimb2;

        datax = _datax;
        datay = _datay;
        dataz = _dataz;

        thrust::host_vector<T> _cdatax, _cdatay, _cdataz;
        if (useTrilinearCorrection)
        {
            par2::cornerfield::build(grid, _cdatax);
            par2::cornerfield::build(grid, _cdatay);
            par2::cornerfield::build(grid, _cdataz);

            par2::cornerfield::computeCornerVelocities(grid, _datax, _datay, _dataz,
                                                    _cdatax, _cdatay, _cdataz);
        }
        else
        {
            par2::cellfield::build(grid, _cdatax);
            par2::cellfield::build(grid, _cdatay);
            par2::cellfield::build(grid, _cdataz);

            par2::cellfield::computeDriftCorrection(grid, _datax, _datay, _dataz,
                                                    _cdatax, _cdatay, _cdataz,
                                                    molecularDiffusion, alphaL, alphaT);
        }
        cdatax = _cdatax;
        cdatay = _cdatay;
        cdataz = _cdataz;

        states.resize(maxParticles);
        thrust::counting_iterator<unsigned int> count(0);
        thrust::for_each(count, count+maxParticles, InitCURAND<T>(_seed, states));

        moveParticle.initialize(datax,
                                datay,
                                dataz,
                                cdatax,
                                cdatay,
                                cdataz,
                                mmb1,
                                mmb2,
                                mimb1,
                                mimb2,
                                nParticles,
                                molecularDiffusion,
                                alphaL,
                                alphaT,
                                massType,
                                mass1alpha,
                                mass1beta,
                                mass2alpha,
                                mass2beta,
                                states,
                                useTrilinearCorrection);

        hipDeviceSynchronize();
    }

    template<typename T>
    unsigned int PParticles<T>::size() const
    {
        return nParticles;
    }

    template<typename T>
    void PParticles<T>::initializeBox(T p1x, T p1y, T p1z,
                                      T p2x, T p2y, T p2z)
    {
        thrust::counting_iterator<unsigned int> count(0);
        auto pBeg = thrust::make_zip_iterator(
            thrust::make_tuple(cx.begin(), cy.begin(), cz.begin(), cp.begin()));

        auto functor = InitVolume<T>(states, grid, mmb1, nParticles, p1x, p1y, p1z, p2x, p2y, p2z);

        for (auto i = 0; i*maxParticles < nParticles; i++)
        {
            unsigned int kernelSize = maxParticles;
            if (kernelSize > nParticles - i*maxParticles)
            {
                kernelSize = nParticles - i*maxParticles;
            }
            thrust::transform(count,
                              count + kernelSize,
                              pBeg + i*maxParticles,
                              functor);
        }
        hipDeviceSynchronize();
    }

    template<typename T>
    void PParticles<T>::move(T dt)
    {
        thrust::counting_iterator<unsigned int> count(0);
        moveParticle.setTimeStep(dt);
        thrust::copy(mmb1.begin(), mmb1.end(), mmb2.begin());
        thrust::copy(mimb1.begin(), mimb1.end(), mimb2.begin());

        for (auto i = 0; i*maxParticles < nParticles; i++)
        {
            unsigned int kernelSize = maxParticles;
            if (kernelSize > nParticles - i*maxParticles)
            {
                kernelSize = nParticles - i*maxParticles;
            }

            auto pBeg = thrust::make_zip_iterator(
                thrust::make_tuple(cx.begin() + i*maxParticles,
                                   cy.begin() + i*maxParticles,
                                   cz.begin() + i*maxParticles,
                                   cp.begin() + i*maxParticles,
                                   cm.begin() + i*maxParticles,
                                   count));
            //auto pEnd = thrust::make_zip_iterator(
            //    thrust::make_tuple(cx.end(),   cy.end()  , cz.end()  , count+kernelSize));

            thrust::transform(pBeg, pBeg + kernelSize, pBeg, moveParticle);
        }
        hipDeviceSynchronize();

    }

    template<typename T>
    void PParticles<T>::exportCSV(const std::string &fileName) const
    {
        // Copy to host memory
        thrust::host_vector<T> hx = cx;
        thrust::host_vector<T> hy = cy;
        thrust::host_vector<T> hz = cz;
        thrust::host_vector<int> hp = cp;
        thrust::host_vector<T> hm = cm;

        std::ofstream outStream;
        outStream.open(fileName);
        if (outStream.is_open())
        {
            outStream << "id,x coord,y coord,z coord,mass phase,probability of mobilephase" << std::endl;
            for (unsigned int i = 0; i < nParticles; i++)
            {
                outStream << i << "," << hx[i] << "," << hy[i] << "," << hz[i] << "," << hp[i] << "," << hm[i]
                          << std::endl;
            }
        }
        else
        {
            throw std::runtime_error(std::string("Could not open file ") + fileName);
        }
        outStream.close();
    }

    template<typename T>
    void PParticles<T>::massexportCSV(const std::string &fileName) const
    {
        // Copy to host memory
        thrust::host_vector<int> hmmb1 = mmb1;
        thrust::host_vector<int> hmimb1 = mimb1;
        int nt = hmmb1.size();

        std::ofstream outStream;
        outStream.open(fileName);
        if (outStream.is_open())
        {
            outStream << "grid_id, mmb, mimb" << std::endl;
            for (int i = 0; i < nt; i++)
            {
                outStream << i << "," << T(hmmb1[i])/T(nParticles) << "," << T(hmimb1[i])/T(nParticles) << std::endl;
            }
        }
        else
        {
            throw std::runtime_error(std::string("Could not open file ") + fileName);
        }
        outStream.close();
    }

    template<typename T>
    struct isInside
    {
        T plane;

        T p1x, p1y, p1z;
        T p2x, p2y, p2z;
        isInside(T _p1x, T _p1y, T _p1z,
                 T _p2x, T _p2y, T _p2z)
        {
            p1x = _p1x;
            p1y = _p1y;
            p1z = _p1z;
            p2x = _p2x;
            p2y = _p2y;
            p2z = _p2z;
        }

        using Position = thrust::tuple<T, T, T>;

        __device__
        bool operator()(Position p) const
        {
            return (p1x <= thrust::get<0>(p) && thrust::get<0>(p) <= p2x) &&
                   (p1y <= thrust::get<1>(p) && thrust::get<1>(p) <= p2y) &&
                   (p1z <= thrust::get<2>(p) && thrust::get<2>(p) <= p2z);
        }
    };

    template<typename T>
    T PParticles<T>::concentrationBox(T p1x, T p1y, T p1z,
                                      T p2x, T p2y, T p2z) const
    {
        auto pBeg = thrust::make_zip_iterator(
            thrust::make_tuple(cx.begin(), cy.begin(), cz.begin()));
        auto pEnd = thrust::make_zip_iterator(
            thrust::make_tuple(cx.end(),   cy.end()  , cz.end()  ));

        return thrust::count_if(pBeg, pEnd,
                    isInside<T>(p1x, p1y, p1z, p2x, p2y, p2z))/T(nParticles);
    }

    template<typename T>
    struct isAfter
    {
        T plane;

        isAfter(T _plane) : plane(_plane) {};

        __device__
        bool operator()(T x)
        {
            return x > plane;
        }
    };

    template<typename T>
    T PParticles<T>::concentrationAfterX(T xplane) const
    {
        return thrust::count_if(cx.begin(), cx.end(),
                    isAfter<T>(xplane))/T(nParticles);
    }

}
